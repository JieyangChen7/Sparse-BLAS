#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <vector>
#include <iostream>
#include <cstdio>
#include <pthread.h>
#include "spmv_task.h"
#include "spmv_kernel.h"
#include <omp.h>

using namespace std;

void * spmv_worker(void * arg);

void generate_tasks(int m, int n, int nnz, double * alpha,
				    double * csrVal, int * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	int nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr);

void assign_task(spmv_task * t, int dev_id, hipStream_t stream);

void run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel);

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream);

void print_task_info(spmv_task * t);

struct pthread_arg_struct
{
	vector<spmv_task *> * arg_spmv_task_pool;
	vector<spmv_task *> * arg_spmv_task_completed;
	int arg_dev_id;
};


int spMV_mgpu_v2(int m, int n, int nnz, double * alpha,
				  double * csrVal, int * csrRowPtr, int * csrColIndex, 
				  double * x, double * beta,
				  double * y,
				  int ngpu, 
				  int kernel,
				  int nb)
{
	vector<spmv_task *> spmv_task_pool;
	vector<spmv_task *> spmv_task_completed;

	generate_tasks(m, n, nnz, alpha,
				  csrVal, csrRowPtr, csrColIndex, 
				  x, beta, y, nb,
				  &spmv_task_pool);

	// pthread_arg_struct * arg1 = new pthread_arg_struct();
	// arg1->arg_spmv_task_pool = &spmv_task_pool;
	// arg1->arg_spmv_task_completed = &spmv_task_completed;
	// arg1->arg_dev_id = 0;

	// omp_set_num_threads(ngpu);
	// #pragma omp parallel default (shared)
	// {
	// 	int c;
	// 	unsigned int dev_id = omp_get_thread_num();
	// 	hipSetDevice(dev_id);
		

	// 	int copy_of_workspace = 2;

	// 	hipsparseStatus_t status[copy_of_workspace];
	// 	hipStream_t stream[copy_of_workspace];
	// 	hipsparseHandle_t handle[copy_of_workspace];



	// 	double ** dev_csrVal = new double * [copy_of_workspace];
	// 	int ** dev_csrRowPtr = new int    * [copy_of_workspace];
	// 	int ** dev_csrColIndex = new int  * [copy_of_workspace];
	// 	double ** dev_x = new double      * [copy_of_workspace];
	// 	double ** dev_y = new double      * [copy_of_workspace];

	// 	for (c = 0; c < copy_of_workspace; c++) {
	// 		hipStreamCreate(&(stream[c]));
	// 		status[c] = hipsparseCreate(&(handle[c])); 
	// 		if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
	// 		{ 
	// 			printf("CUSPARSE Library initialization failed");
	// 			//return 1; 
	// 		} 
	// 		status[c] = hipsparseSetStream(handle[c], stream[c]);
	// 		if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
	// 		{ 
	// 			printf("Stream bindind failed");
	// 			//return 1;
	// 		} 

	// 		hipMalloc((void**)&(dev_csrVal[c]),      nnz      * sizeof(double));
	// 		hipMalloc((void**)&(dev_csrRowPtr[c]),   (m + 1) * sizeof(int)   );
	// 		hipMalloc((void**)&(dev_csrColIndex[c]), nnz      * sizeof(int)   );
	// 		hipMalloc((void**)&(dev_x[c]),           n       * sizeof(double));
	//     	hipMalloc((void**)&(dev_y[c]),           m       * sizeof(double));

 //    	}

 //    	c = 0; 
    
	// 	while (true) {

	// 		spmv_task * curr_spmv_task;

	// 		for (c = 0; c < copy_of_workspace; c++) {


	// 			#pragma omp critical
	// 			{
	// 				if(spmv_task_pool.size() > 0) {
	// 					curr_spmv_task = spmv_task_pool[spmv_task_pool.size() - 1];
	// 					spmv_task_pool.pop_back();
	// 				} else {
	// 					curr_spmv_task = NULL;
	// 				}
	// 			}

	// 			if (curr_spmv_task) {

	// 				curr_spmv_task->dev_csrVal = dev_csrVal[c];
	// 				curr_spmv_task->dev_csrRowPtr = dev_csrRowPtr[c];
	// 				curr_spmv_task->dev_csrColIndex = dev_csrColIndex[c];
	// 				curr_spmv_task->dev_x = dev_x[c];
	// 				curr_spmv_task->dev_y = dev_y[c];
	// 				assign_task(curr_spmv_task, dev_id, stream[c]);
	// 				run_task(curr_spmv_task, dev_id, handle[c], kernel);
	// 				finalize_task(curr_spmv_task, dev_id, stream[c]);
	// 			}
	// 		}
	// 		if (!curr_spmv_task) {
	// 			break;
	// 		}
	// 	}

	// 	hipDeviceSynchronize();

	// 	for (c = 0; c < copy_of_workspace; c++) {

	// 		hipFree(dev_csrVal[c]);
	// 		hipFree(dev_csrRowPtr[c]);
	// 		hipFree(dev_csrColIndex[c]);
	// 		hipFree(dev_x[c]);
	// 		hipFree(dev_y[c]);
	// 	}
	// }
}



void generate_tasks(int m, int n, int nnz, double * alpha,
				    double * csrVal, int * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	int nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr) {

	int num_of_tasks = (nnz + nb - 1) / nb;
	cout << "num_of_tasks = " << num_of_tasks << endl;

	int curr_row;
	int t;
	int d;

	spmv_task * spmv_task_pool = new spmv_task[num_of_tasks];

	// Calculate the start and end index
	for (t = 0; t < num_of_tasks; t++) {
		long long tmp1 = t * (long long)nnz;
		long long tmp2 = (t + 1) * (long long)nnz;

		double tmp3 = (double)(tmp1 / num_of_tasks);
		double tmp4 = (double)(tmp2 / num_of_tasks);

		// cout << "tmp1 = " << tmp1 << endl;
		// cout << "tmp2 = " << tmp2 << endl;

		// cout << "tmp3 = " << tmp3 << endl;
		// cout << "tmp4 = " << tmp4 << endl;

		spmv_task_pool[t].start_idx = floor((double)(tmp1 / num_of_tasks));
		spmv_task_pool[t].end_idx   = floor((double)(tmp2 / num_of_tasks)) - 1;
		spmv_task_pool[t].dev_nnz = spmv_task_pool[t].end_idx - spmv_task_pool[t].start_idx + 1;

		// cout << "spmv_task_pool[t].start_idx = " << spmv_task_pool[t].start_idx << endl;
		// cout << "spmv_task_pool[t].end_idx = " << spmv_task_pool[t].end_idx << endl; 
	}

	// Calculate the start and end row
	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {

		spmv_task_pool[t].start_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].start_idx);
		//cout << "spmv_task_pool[t].start_row = " << spmv_task_pool[t].start_row << endl;
		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].start_idx > csrRowPtr[spmv_task_pool[t].start_row]) {
			spmv_task_pool[t].start_flag = true;
			spmv_task_pool[t].y2 = y[spmv_task_pool[t].start_row];
		} else {
			spmv_task_pool[t].start_flag = false;
		}
	}

	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].end_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].end_idx);
		//cout << "spmv_task_pool[t].end_row = " << spmv_task_pool[t].end_row << endl;

		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].end_idx < csrRowPtr[spmv_task_pool[t].end_row + 1] - 1)  {
			spmv_task_pool[t].end_flag = true;
		} else {
			spmv_task_pool[t].end_flag = false;
		}
	}

	// Cacluclate dimensions
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].dev_m = spmv_task_pool[t].end_row - spmv_task_pool[t].start_row + 1;
		spmv_task_pool[t].dev_n = n;
		// cout << "spmv_task_pool[t].start_idx = " << spmv_task_pool[t].start_idx << endl;
		// cout << "spmv_task_pool[t].end_idx = " << spmv_task_pool[t].end_idx << endl; 
		// cout << "spmv_task_pool[t].start_row = " << spmv_task_pool[t].start_row << endl;
		// cout << "spmv_task_pool[t].end_row = " << spmv_task_pool[t].end_row << endl;
		// cout << "spmv_task_pool[t].dev_m = " << spmv_task_pool[t].dev_m << endl;
	}

	for (t = 0; t < num_of_tasks; t++) {


		//cout << "spmv_task_pool[t].dev_m + 1 = " << spmv_task_pool[t].dev_m + 1 << endl;
		spmv_task_pool[t].host_csrRowPtr = new int [spmv_task_pool[t].dev_m + 1];
		spmv_task_pool[t].host_csrRowPtr[0] = 0;
		spmv_task_pool[t].host_csrRowPtr[spmv_task_pool[t].dev_m] = spmv_task_pool[t].dev_nnz;

		memcpy(&(spmv_task_pool[t].host_csrRowPtr[1]), 
			   &csrRowPtr[spmv_task_pool[t].start_row + 1], 
			   (spmv_task_pool[t].dev_m - 1) * sizeof(int) );

		for (int j = 1; j < spmv_task_pool[t].dev_m; j++) {
			spmv_task_pool[t].host_csrRowPtr[j] -= spmv_task_pool[t].start_idx;
		}

		spmv_task_pool[t].host_csrColIndex = csrColIndex;
		spmv_task_pool[t].host_csrVal = csrVal;
		spmv_task_pool[t].host_y = y;
		spmv_task_pool[t].host_x = x;
		spmv_task_pool[t].local_result_y = new double[spmv_task_pool[t].dev_m];
		spmv_task_pool[t].alpha = new double[1];
		spmv_task_pool[t].beta = new double[1]; 
		spmv_task_pool[t].alpha[0] = *alpha;
		spmv_task_pool[t].beta[0] = *beta;

	}

	for (t = 0; t < num_of_tasks; t++) {
		hipsparseStatus_t status = hipsparseCreateMatDescr(&(spmv_task_pool[t].descr));
		if (status != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Matrix descriptor initialization failed");
			//return 1;
		} 	
		hipsparseSetMatType(spmv_task_pool[t].descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
		hipsparseSetMatIndexBase(spmv_task_pool[t].descr,HIPSPARSE_INDEX_BASE_ZERO);
	}

	for (t = 0; t < num_of_tasks; t++) {
		(*spmv_task_pool_ptr).push_back(&spmv_task_pool[t]);
	}

}

void assign_task(spmv_task * t, int dev_id, hipStream_t stream){
	t->dev_id = dev_id;
	// hipSetDevice(dev_id);
	// hipMalloc((void**)&(t->dev_csrVal),      (t->dev_nnz)   * sizeof(double));
	// hipMalloc((void**)&(t->dev_csrRowPtr),   (t->dev_m + 1) * sizeof(int)   );
	// hipMalloc((void**)&(t->dev_csrColIndex), (t->dev_nnz)   * sizeof(int)   );
	// hipMalloc((void**)&(t->dev_x),           (t->dev_n)     * sizeof(double));
 //    hipMalloc((void**)&(t->dev_y),           (t->dev_m)     * sizeof(double));

    hipMemcpyAsync(t->dev_csrRowPtr,   t->host_csrRowPtr,          
    			   (size_t)((t->dev_m + 1) * sizeof(int)), hipMemcpyHostToDevice, stream);

	hipMemcpyAsync(t->dev_csrColIndex, &(t->host_csrColIndex[t->start_idx]), 
		           (size_t)(t->dev_nnz * sizeof(int)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_csrVal,      &(t->host_csrVal[t->start_idx]),
		           (size_t)(t->dev_nnz * sizeof(double)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_y, &(t->host_y[t->start_row]), 
		           (size_t)(t->dev_m * sizeof(double)), hipMemcpyHostToDevice, stream); 
	
	hipMemcpyAsync(t->dev_x, t->host_x,
				   (size_t)(t->dev_n * sizeof(double)),  hipMemcpyHostToDevice, stream); 
}

void run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel){
	//hipSetDevice(dev_id);

	hipStream_t stream;

	hipsparseGetStream(handle, &stream);

	cout << "dev_m[d] = " << t->dev_m << endl;
	cout << "dev_n[d] = " << t->dev_n << endl;
	hipMemcpyAsync( t->host_csrRowPtr,  t->dev_csrRowPtr, (size_t)(( t->dev_m + 1) * sizeof(int)), hipMemcpyDeviceToHost, stream);
	hipMemcpyAsync(&(t->host_csrColIndex[t->start_idx]),  t->dev_csrColIndex,  (size_t)( t->dev_nnz * sizeof(int)),     hipMemcpyDeviceToHost, stream); 
	hipMemcpyAsync(&(t->host_csrVal[t->start_idx]),  t->dev_csrVal,            (size_t)( t->dev_nnz * sizeof(double)),  hipMemcpyDeviceToHost, stream); 

	hipMemcpyAsync(&(t->host_y[t->start_row]),  t->dev_y,  (size_t)( t->dev_m*sizeof(double)),  hipMemcpyDeviceToHost, stream); 
	hipMemcpyAsync(t->host_x, t->dev_x,                (size_t)(t->dev_n*sizeof(double)),  hipMemcpyDeviceToHost, stream); 

	hipDeviceSynchronize();

	cout << "dev_csrRowPtr = [";
	for (int i = 0; i < t->dev_m + 1; i++) {
		cout << t->host_csrRowPtr[i] << ", ";
	}
	cout << "]" << endl;
	cout << "csrColIndex = [";
	for (int i = 0; i < t->dev_nnz; i++) {
		cout << t->host_csrColIndex[t->start_idx+i] << ", ";
	}
	cout << "]" << endl;
	cout << "csrVal[start_idx[d]] = [";
	for (int i = 0; i < t->dev_nnz; i++) {
		cout << t->host_csrVal[t->start_idx+i] << ", ";
	}
	cout << "]" << endl;
	cout << "y[start_row[d]] = [";
	for (int i = 0; i < t->dev_m; i++) {
		cout << t->host_y[t->start_row+i] << ", ";
	}
	cout << "]" << endl;
	cout << "dev_x[d] = [";
	for (int i = 0; i < t->dev_n; i++) {
		cout << t->host_x[i] << ", ";
	}
	cout << "]" << endl;

	cout << "t->alpha = " << *(t->alpha) << endl;
	cout << "t->beta = " << *(t->beta) << endl;

	hipsparseStatus_t status;
	if(kernel == 1) {
		status = hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
								t->dev_m, t->dev_n, t->dev_nnz, 
								t->alpha, t->descr, t->dev_csrVal, 
								t->dev_csrRowPtr, t->dev_csrColIndex, 
								t->dev_x,  t->beta, t->dev_y); 
	} else if (kernel == 2) {
		status = cusparseDcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
									t->dev_m, t->dev_n, t->dev_nnz, 
									t->alpha, t->descr, t->dev_csrVal, 
									t->dev_csrRowPtr, t->dev_csrColIndex, 
									t->dev_x,  t->beta, t->dev_y); 
	} else if (kernel == 3) {
		// int err = 0;
		// anonymouslibHandle<int, unsigned int, double> A(t->dev_m, t->dev_n);
		// err = A.inputCSR(
		// 	            t->dev_nnz, 
		// 				t->dev_csrRowPtr, 
		// 				t->dev_csrColIndex, 
		// 				t->dev_csrVal);
		// //cout << "inputCSR err = " << err << endl;
		// err = A.setX(t->dev_x);
		// //cout << "setX err = " << err << endl;
		// A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);
		// A.warmup();
		// err = A.asCSR5();
		// //cout << "asCSR5 err = " << err << endl;
		// err = A.spmv(*(t->alpha), t->dev_y);
	}

}

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream) {
	//hipSetDevice(dev_id);

	hipMemcpyAsync(t->local_result_y,   t->dev_y,          
    			   (size_t)((t->dev_m) * sizeof(double)), 
    			   hipMemcpyDeviceToHost, stream);
	// hipFree(t->dev_csrVal);
	// hipFree(t->dev_csrRowPtr);
	// hipFree(t->dev_csrColIndex);
	// hipFree(t->dev_x);
}

void print_task_info(spmv_task * t) {
	cout << "start_idx = " << t->start_idx << endl;
	cout << "end_idx = " << t->end_idx << endl;
}
