#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
using namespace std;

int spMV_mgpu_baseline(int m, int n, int nnz, double * alpha,
				 double * csrVal, int * csrRowPtr, int * csrColIndex, 
				 double * x, double * beta,
				 double * y,
				 int ngpu,
				 double * time_parse,
				 double * time_comm,
				 double * time_comp,
				 double * time_post){

	double curr_time = 0.0;
	double tmp = 0.0;

	curr_time = get_time();


	hipStream_t * stream = new hipStream_t [ngpu];

	hipError_t * cudaStat1 = new hipError_t[ngpu];
	hipError_t * cudaStat2 = new hipError_t[ngpu];
	hipError_t * cudaStat3 = new hipError_t[ngpu];
	hipError_t * cudaStat4 = new hipError_t[ngpu];
	hipError_t * cudaStat5 = new hipError_t[ngpu];
	hipError_t * cudaStat6 = new hipError_t[ngpu];

	hipsparseStatus_t * status = new hipsparseStatus_t[ngpu];
	hipsparseHandle_t * handle = new hipsparseHandle_t[ngpu];
	hipsparseMatDescr_t * descr = new hipsparseMatDescr_t[ngpu];

	int  * start_row  = new int[ngpu];
	int  * end_row    = new int[ngpu];
		
	int * dev_m            = new int      [ngpu];
	int * dev_n            = new int      [ngpu];
	int * dev_nnz          = new int      [ngpu];
	int ** host_csrRowPtr  = new int    * [ngpu];
	int ** dev_csrRowPtr   = new int    * [ngpu];
	int ** dev_csrColIndex = new int    * [ngpu];
	double ** dev_csrVal   = new double * [ngpu];


	double ** dev_x = new double * [ngpu];
	double ** dev_y = new double * [ngpu];

	for (int d = 0; d < ngpu; d++){

		hipSetDevice(d);

		cout << "GPU " << d << ":" << endl;
		start_row[d] = floor((d)     * m / ngpu);
		end_row[d]   = floor((d + 1) * m / ngpu) - 1;

		cout << "start_row: " << start_row[d] << ", " << "end_row: "<< end_row[d] << endl;

		dev_m[d]   = end_row[d] - start_row[d] + 1;
		dev_n[d]   = n;
		dev_nnz[d] = csrRowPtr[end_row[d] + 1] - csrRowPtr[start_row[d]];

		cout << "dev_nnz[d] = " << dev_nnz[d] << " = " << csrRowPtr[end_row[d] + 1] << " - " << csrRowPtr[start_row[d]] << endl;

		cout << "dev_m[d]: " << dev_m[d] << ", dev_n[d]: " << dev_n[d] << ", dev_nnz[d]: " << dev_nnz[d] << endl;

		host_csrRowPtr[d] = new int[dev_m[d] + 1];

		memcpy((void *)host_csrRowPtr[d], 
			   (void *)&csrRowPtr[start_row[d]], 
			   (dev_m[d] + 1) * sizeof(int));

		cout << "csrRowPtr (before): ";
		for (int i = 0; i <= dev_m[d]; i++) {
			cout << host_csrRowPtr[d][i] << ", ";
		}
		cout << endl;

		for (int i = 0; i < dev_m[d] + 1; i++) {
			host_csrRowPtr[d][i] -= csrRowPtr[start_row[d]];
		}

		cout << "csrRowPtr (after): ";
		for (int i = 0; i <= dev_m[d]; i++) {
			cout << host_csrRowPtr[d][i] << ", ";
		}
		cout << endl;

	}


	*time_parse = get_time() - curr_time;
	curr_time = get_time();

	for (int d = 0; d < ngpu; d++){
		hipSetDevice(d);

		hipStreamCreate(&(stream[d]));
		
		status[d] = hipsparseCreate(&(handle[d])); 
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("CUSPARSE Library initialization failed");
			return 1; 
		} 
		status[d] = hipsparseSetStream(handle[d], stream[d]);
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Stream bindind failed");
			return 1;
		} 
		status[d] = hipsparseCreateMatDescr(&descr[d]);
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Matrix descriptor initialization failed");
			return 1;
		} 	
		hipsparseSetMatType(descr[d],HIPSPARSE_MATRIX_TYPE_GENERAL); 
		hipsparseSetMatIndexBase(descr[d],HIPSPARSE_INDEX_BASE_ZERO); 

		cudaStat1[d] = hipMalloc((void**)&dev_csrRowPtr[d],   (dev_m[d] + 1) * sizeof(int));
		cudaStat2[d] = hipMalloc((void**)&dev_csrColIndex[d],100 * sizeof(int)); 
		cudaStat3[d] = hipMalloc((void**)&dev_csrVal[d],      dev_nnz[d] * sizeof(double)); 

		cudaStat4[d] = hipMalloc((void**)&dev_x[d],           dev_n[d] * sizeof(double)); 
		cudaStat5[d] = hipMalloc((void**)&dev_y[d],           dev_m[d] * sizeof(double)); 
		

		if ((cudaStat1[d] != hipSuccess) || 
			(cudaStat2[d] != hipSuccess) || 
			(cudaStat3[d] != hipSuccess) || 
			(cudaStat4[d] != hipSuccess) || 
			(cudaStat5[d] != hipSuccess)) 
		{ 
			printf("Device malloc failed");
			return 1; 
		} 

		cout << "Start copy to GPUs...";
		cudaStat1[d] = hipMemcpy(dev_csrRowPtr[d],   host_csrRowPtr[d],                  (size_t)((dev_m[d] + 1) * sizeof(int)), hipMemcpyHostToDevice);
		if (cudaStat1[d] != hipSuccess) cout << "error 1" << endl;
		cout << "host_csrRowPtr[d] = ";
		for (int i = 0; i < dev_m[d] + 1; ++i)
		{
			cout << host_csrRowPtr[d][i] << ", ";
		}
		cout << endl;
		//cudaStat2[d] = hipMemcpy(dev_csrColIndex[d], &csrColIndex[csrRowPtr[start_row[d]]], (size_t)(dev_nnz[d] * sizeof(int)),   hipMemcpyHostToDevice); 
		for (int i = 0 ; i<dev_nnz[d]; i+=1) {
			cudaStat2[d] = hipMemcpy(dev_csrColIndex[d], &csrColIndex[csrRowPtr[start_row[d]]], i*sizeof(int),   hipMemcpyHostToDevice); 
			
			if (cudaStat2[d] != hipSuccess) cout << "i=" << i <<" error 2 " << cudaStat2[d] <<  endl;
		}
		cout << "csrColIndex[d] = ";
		for (int i = 0; i < dev_nnz[d]; ++i)
		{
			cout << csrColIndex[csrRowPtr[start_row[d]]+i] << ", ";
		}
		cout << endl;
		cudaStat3[d] = hipMemcpy(dev_csrVal[d],      &csrVal[csrRowPtr[start_row[d]]],      (size_t)(dev_nnz[d] * sizeof(double)), hipMemcpyHostToDevice);
		if (cudaStat3[d] != hipSuccess) cout << "error 3 " << cudaStat3[d] <<  endl; 

		cout << "csrVal[d] = ";
		for (int i = 0; i < dev_nnz[d]; ++i)
		{
			cout << csrVal[csrRowPtr[start_row[d]]+i] << ", ";
		}
		cout << endl;


		cudaStat4[d] = hipMemcpy(dev_y[d], &y[start_row[d]], (size_t)(dev_m[d]*sizeof(double)), hipMemcpyHostToDevice); 
		if (cudaStat4[d] != hipSuccess) cout << "error 4" << endl;

		cudaStat5[d] = hipMemcpy(dev_x[d], x,                (size_t)(dev_n[d]*sizeof(double)), hipMemcpyHostToDevice); 
		if (cudaStat5[d] != hipSuccess) cout << "error 5" << endl;

		// cout << "x = ";
		// for (int i = 0; i < dev_n[d]; ++i)
		// {
		// 	cout << x[i] << ", ";
		// }
		// cout << endl;

		// cout << "y = ";
		// for (int i = 0; i < dev_m[d]; ++i)
		// {
		// 	cout << y[i] << ", ";
		// }
		// cout << endl;

		if ((cudaStat1[d] != hipSuccess) ||
		 	(cudaStat2[d] != hipSuccess) ||
		  	(cudaStat3[d] != hipSuccess) ||
		   	(cudaStat4[d] != hipSuccess) ||
		    (cudaStat5[d] != hipSuccess)) 
		{ 
			printf("Memcpy from Host to Device failed"); 
			//return 1; 
		} 

	}

	*time_comm = get_time() - curr_time;
	curr_time = get_time();


	cout << "Start computation ... " << endl;
	 int repeat_test = 1;
	 double start = get_time();
	 for (int i = 0; i < repeat_test; i++) 
	 {
		for (int d = 0; d < ngpu; ++d) 
		{
			//tmp = get_time();
			hipSetDevice(d);
			//cout << "dev_m[d]: " << dev_m[d] << ", dev_n[d]: " << dev_n[d] << ", dev_nnz[d]: " << dev_nnz[d] << endl;
			status[d] = hipsparseDcsrmv(handle[d],HIPSPARSE_OPERATION_NON_TRANSPOSE, 
									   dev_m[d], dev_n[d], dev_nnz[d], 
									   alpha, descr[d], dev_csrVal[d], 
									   dev_csrRowPtr[d], dev_csrColIndex[d], 
									   dev_x[d], beta, dev_y[d]);		 	
			// hipDeviceSynchronize();
			// cout << "computation " << d << " : " << get_time()-tmp << endl;
			
		 	
		}
		for (int d = 0; d < ngpu; ++d) 
		{
			hipSetDevice(d);
			hipDeviceSynchronize();
		}


	}

	*time_comp = get_time() - curr_time;
	curr_time = get_time();

	for (int d = 0; d < ngpu; d++)
	{
		hipMemcpy( &y[start_row[d]], dev_y[d], (size_t)(dev_m[d]*sizeof(double)),  hipMemcpyDeviceToHost);
	}

	


	// double end = get_time();
	// double time = end - start;
	// printf("spMV_mgpu_v1 time = %f s\n", time);	
	// long long flop = nnz * 2;
	// flop *= repeat_test;
	// double gflop = (double)flop/1e9;
	// printf("gflop = %f\n", gflop);
	// double gflops = gflop / time;
	// printf("GFLOPS = %f\n", gflops);
	// return gflops;

	for (int d = 0; d < ngpu; d++) {
		hipSetDevice(d);
		hipFree(dev_csrVal[d]);
		hipFree(dev_csrRowPtr[d]);
		hipFree(dev_csrColIndex[d]);
		hipFree(dev_x[d]);
		hipFree(dev_y[d]);
	}

	*time_post = get_time() - curr_time;
	// 	delete[] dev_csrVal;
	// 	delete[] dev_csrRowPtr;
	// 	delete[] dev_csrColIndex;
	// 	delete[] dev_x;
	// 	delete[] dev_y;
	// 	delete[] host_csrRowPtr;
	// 	delete[] start_row;
	// 	delete[] end_row;
		
	

}