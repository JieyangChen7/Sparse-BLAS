#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include "mmio.h"
#include <float.h>
#include <omp.h>
//#include "anonymouslib_cuda.h"
#include <hip/hip_runtime_api.h>
#include "spmv_kernel.h"
using namespace std;



void print_error(hipsparseStatus_t status) {
	if (status == HIPSPARSE_STATUS_NOT_INITIALIZED)
		cout << "HIPSPARSE_STATUS_NOT_INITIALIZED" << endl;
	else if (status == HIPSPARSE_STATUS_ALLOC_FAILED)
		cout << "HIPSPARSE_STATUS_ALLOC_FAILED" << endl;
	else if (status == HIPSPARSE_STATUS_INVALID_VALUE)
		cout << "HIPSPARSE_STATUS_INVALID_VALUE" << endl;
	else if (status == HIPSPARSE_STATUS_ARCH_MISMATCH)
		cout << "HIPSPARSE_STATUS_ARCH_MISMATCH" << endl;
	else if (status == HIPSPARSE_STATUS_INTERNAL_ERROR)
		cout << "HIPSPARSE_STATUS_INTERNAL_ERROR" << endl;
	else if (status == HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED)
		cout << "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED" << endl;
}


int main(int argc, char *argv[]) {


	// omp_set_num_threads(8);
	// cout << "omp_get_max_threads = " << omp_get_max_threads() << endl;
	// cout << "omp_get_thread_limit = " << omp_get_thread_limit() << endl;
	// #pragma omp parallel// default (shared)
	// {
	// 	cout << "omp_get_num_threads = " << omp_get_num_threads() << endl;
	// 	cout << "omp_get_max_threads = " << omp_get_max_threads() << endl;
	// 	cout << "omp_get_thread_limit = " << omp_get_thread_limit() << endl;


	// }

	if (argc < 6) {
		cout << "Incorrect number of arguments!" << endl;
		cout << "Usage ./spmv [input matrix file] [number of GPU(s)] [number of test(s)] [kernel version (1-3)] [data type ('f' or 'b')]"  << endl;
		return -1;
	}

	char input_type = argv[1][0];

	char * filename = argv[2];

	int ngpu = atoi(argv[3]);
	int repeat_test = atoi(argv[4]);
	int kernel_version = atoi(argv[5]);
	char data_type = argv[6][0];
	int divide = atoi(argv[7]);
	int copy_of_workspace = atoi(argv[8]);

	int ret_code;
    MM_typecode matcode;
    FILE *f;
    int m, n, nnz;   
    int * cooRowIndex;
    int * cooColIndex;
    double * cooVal;
    int * csrRowPtr;

    if (input_type == 'f') {

	    cout << "loading input matrix from " << filename << endl;
	    if ((f = fopen(filename, "r")) == NULL) {
	        exit(1);
	    }
	    if (mm_read_banner(f, &matcode) != 0) {
	        printf("Could not process Matrix Market banner.\n");
	        exit(1);
	    }
	    if ((ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnz)) !=0) {
	        exit(1);
	    }
	    cout << "m: " << m << " n: " << n << " nnz: " << nnz << endl;

	    //cooRowIndex = (int *) malloc(nnz * sizeof(int));
	    //cooColIndex = (int *) malloc(nnz * sizeof(int));
	    //cooVal      = (double *) malloc(nnz * sizeof(double));

	    hipHostMalloc((void **)&cooRowIndex, nnz * sizeof(int));
	    hipHostMalloc((void **)&cooColIndex, nnz * sizeof(int));
	    hipHostMalloc((void **)&cooVal, nnz * sizeof(double));
	   
	    // Read matrix from file into COO format
	    for (int i = 0; i < nnz; i++) {
	    	if (data_type == 'b') { // binary input
	    		fscanf(f, "%d %d\n", &cooRowIndex[i], &cooColIndex[i]);
	    		cooVal[i] = 0.00001;

	    	} else if (data_type == 'f'){ // float input
	        	fscanf(f, "%d %d %lg\n", &cooRowIndex[i], &cooColIndex[i], &cooVal[i]);
	        }
	        cooRowIndex[i]--;  
	        cooColIndex[i]--;

	        if (cooRowIndex[i] < 0 || cooColIndex[i] < 0) { // report error
	       		cout << "i = " << i << " [" <<cooRowIndex[i] << ", " << cooColIndex[i] << "] = " << cooVal[i] << endl;
	       	}
		}
	} else if(input_type == 'g') { // generate data
		//int n = 10000;
		n = atoi(filename);

		m = n;
		int nb = m / 8;
		double r;
		double r1 = 0.9;
		double r2 = 0.1;

		int p = 0;

		for (int i = 0; i < m; i += nb) {
			if (i == 0) {
				r = r1;
			} else {
				r = r2;
			}
			for (int ii = i; ii < i + nb; ii++) {
				for (int j = 0; j < n * r; j++) {
					p++;
				}
			}
		}


		nnz = p;

		cout << "m: " << m << " n: " << n << " nnz: " << nnz << endl;

		hipHostMalloc((void **)&cooRowIndex, nnz * sizeof(int));
	    hipHostMalloc((void **)&cooColIndex, nnz * sizeof(int));
	    hipHostMalloc((void **)&cooVal, nnz * sizeof(double));

	    p = 0;
		

		cout << "Start generating data ..." << endl;
		for (int i = 0; i < m; i += nb) {
			cout << ((double)p / nnz) * 100 << "%" << endl;
			cout << p << endl;
			if (i == 0) {
				r = r1;
			} else {
				r = r2;
			}
			cout << "Matrix:" << endl;
			for (int ii = i; ii < i + nb; ii++) {
				for (int j = 0; j < n * r; j++) {
					//if (p > nnz) { cout << "error" << endl; break;}
					//else {

					cooRowIndex[p] = ii;
					cooColIndex[p] = j;
					cooVal[p] = 1;//((double) rand() / (RAND_MAX));
					p++;
					cout << 1 << " ";
					//}

				}
				cout << endl;
			}


		}

		cout << "m: " << m << " n: " << n << " nnz: " << p << endl;


		cout << "Done generating data." << endl;


	}



    




	// Convert COO to CSR
    //csrRowPtr = (int *) malloc((m+1) * sizeof(int));
    hipHostMalloc((void **)&csrRowPtr, (m+1) * sizeof(int));

    //cout << "m: " << m << " n: " << n << " nnz: " << nnz << endl;
    long long matrix_data_space = nnz * sizeof(double) + nnz * sizeof(int) + (m+1) * sizeof(int);
    //cout << matrix_data_space << endl;


    cout << "Matrix space size: " << (double)matrix_data_space / 1e9 << " GB." << endl;

    int * counter = new int[m];
    for (int i = 0; i < m; i++) {
    	counter[i] = 0;
    }
	for (int i = 0; i < nnz; i++) {
		counter[cooRowIndex[i]]++;
	}
	//cout << "nnz: " << nnz << endl;
	//cout << "counter: ";
	int t = 0;
	for (int i = 0; i < m; i++) {
		//cout << counter[i] << ", ";
		t += counter[i];
	}
	//cout << t << endl;
	//cout << endl;


	csrRowPtr[0] = 0;
	for (int i = 1; i <= m; i++) {
		csrRowPtr[i] = csrRowPtr[i - 1] + counter[i - 1];
	}

	// cout << "csrRowPtr: ";
	// for (int i = 0; i <= m; i++) {
	// 	cout << csrRowPtr[i] << ", ";
	// }
	// cout << endl;

	double * x;
	double * y1;
	double * y2;
	double * y3;

	//x = (double *)malloc(n * sizeof(double)); 
	//y1 = (double *)malloc(m * sizeof(double)); 
	y2 = (double *)malloc(m * sizeof(double)); 
	//y3 = (double *)malloc(m * sizeof(double)); 

// 	hipHostMalloc((void **)&x, n * sizeof(double));
// 	hipHostMalloc((void **)&y1, m * sizeof(double));
// 	//hipHostMalloc((void **)&y2, m * sizeof(double));
// 	hipHostMalloc((void **)&y3, m * sizeof(double));

// 	for (int i = 0; i < n; i++)
// 	{
// 		x[i] = 1.0;//((double) rand() / (RAND_MAX)); 
// 	}


// 	for (int i = 0; i < m; i++)
// 	{
// 		y1[i] = 0.0;
// 		y2[i] = 0.0;
// 		y3[i] = 0.0;
// 	}



// 	int deviceCount;
// 	hipGetDeviceCount(&deviceCount);
// 	int device;
// 	for (device = 0; device < deviceCount; ++device) 
// 	{
// 	    hipDeviceProp_t deviceProp;
// 	    hipGetDeviceProperties(&deviceProp, device);
// 	    printf("Device %d has compute capability %d.%d.\n",
// 	           device, deviceProp.major, deviceProp.minor);
// 	}

// 	cout << "Using " << ngpu << " GPU(s)." << endl; 

// 	double ALPHA = 1.0;
// 	double BETA = 0.0;

// 	double time_parse = 0.0;
// 	double time_comm = 0.0;
// 	double time_comp = 0.0;
// 	double time_post = 0.0;

// 	double avg_time_parse1 = 0.0;
// 	double avg_time_comm1 = 0.0;
// 	double avg_time_comp1 = 0.0;
// 	double avg_time_post1 = 0.0;

// 	double avg_time_parse2 =  0.0;
// 	double avg_time_comm2 = 0.0;
// 	double avg_time_comp2 = 0.0;
// 	double avg_time_post2 = 0.0;

// 	double avg_time_parse3 =  0.0;
// 	double avg_time_comm3= 0.0;
// 	double avg_time_comp3 = 0.0;
// 	double avg_time_post3 = 0.0;

// 	int warm_up_iter = 1;

// 	//hipProfilerStart();

// 	for (int i = 0; i < repeat_test + warm_up_iter; i++) {
// 		if (i == 0) {
// 			cout << "Warming up GPU(s)..." << endl;
// 		}
// 		if (i == warm_up_iter) {
// 			cout << "Starting tests..." << endl;
// 		}
// 		for (int i = 0; i < m; i++)
// 		{
// 			y1[i] = 0.0;
// 			y2[i] = 0.0;
// 			y3[i] = 0.0;
// 		}

		
// 		time_parse = 0.0;
// 		time_comm = 0.0;
// 		time_comp = 0.0;
// 		time_post = 0.0;

// 		cout << "=============Baseline[start]============" <<endl;

// 		// spMV_mgpu_baseline(m, n, nnz, &ALPHA,
// 		// 			 cooVal, csrRowPtr, cooColIndex, 
// 		// 			 x, &BETA,
// 		// 			 y1,
// 		// 			 ngpu,
// 		// 			 &time_parse,
// 		// 			 &time_comm,
// 		// 			 &time_comp,
// 		// 			 &time_post);
// 		cout << "=============Baseline[done]============" <<endl;

		
	
		
// 		if (i >= warm_up_iter) {
// 			avg_time_parse1 += time_parse;
// 			avg_time_comm1  += time_comm;
// 			avg_time_comp1  += time_comp;
// 			avg_time_post1  += time_post;
// 		}
		
// 		time_parse = 0.0;
// 		time_comm = 0.0;
// 		time_comp = 0.0;
// 		time_post = 0.0;
		
// 		cout << "=============Version 1[start]============" <<endl;

// 		// spMV_mgpu_v1(m, n, nnz, &ALPHA,
// 		// 			 cooVal, csrRowPtr, cooColIndex, 
// 		// 			 x, &BETA,
// 		// 			 y2,
// 		// 			 ngpu,
// 		// 			 &time_parse,
// 		// 			 &time_comm,
// 		// 			 &time_comp,
// 		// 			 &time_post,
// 		// 			 kernel_version);

// 		cout << "=============Version 1[done]============" <<endl;
		
// 		if (i >= warm_up_iter) {
// 			avg_time_parse2 += time_parse;
// 			avg_time_comm2  += time_comm;
// 			avg_time_comp2  += time_comp;
// 			avg_time_post2  += time_post;
// 		}

// 		time_parse = 0.0;
// 		time_comm = 0.0;
// 		time_comp = 0.0;
// 		time_post = 0.0;

// 		cout << "=============Version 2[start]============" <<endl;


// 		//hipProfilerStart();


// 		// spMV_mgpu_v2(m, n, nnz, &ALPHA,
// 		// 			 cooVal, csrRowPtr, cooColIndex, 
// 		// 			 x, &BETA,
// 		// 			 y3,
// 		// 			 ngpu,
// 		// 			 kernel_version,
// 		// 			 ceil(nnz/divide),
// 		// 			 copy_of_workspace,
// 		// 			 &time_parse,
// 		// 			 &time_comp,
// 		// 			 &time_post);

// 		//
// 		cout << "=============Version 2[done]============" <<endl;
		
// 		if (i >= warm_up_iter) {
// 			avg_time_parse3 += time_parse;
// 			avg_time_comm3  += time_comm;
// 			avg_time_comp3  += time_comp;
// 			avg_time_post3  += time_post;
// 		}

	
// 	}

// 	//hipProfilerStop();

	


// 	//cout << "y = [";
// 	bool correct = true;
// 	for(int i = 0; i < m; i++) {
// 		cout << y1[i] << " - "  << y2[i] << " - "<< y3[i] << endl;
// 		if (abs(y1[i] - y3[i]) > 1e-3 ) {
// 			//cout << y1[i] << " - " << y3[i] << endl;
// 			correct = false;
// 		}
// 	}

// 	if (correct) cout << "Pass" << endl;
// 	else cout << "No pass" << endl;
	
	
// 	avg_time_parse1/=repeat_test;
// 	avg_time_comm1/=repeat_test;
// 	avg_time_comp1/=repeat_test;
// 	avg_time_post1/=repeat_test;

// 	avg_time_parse2/=repeat_test;
// 	avg_time_comm2/=repeat_test;
// 	avg_time_comp2/=repeat_test;
// 	avg_time_post2/=repeat_test;

// 	avg_time_parse3/=repeat_test;
// 	avg_time_comm3/=repeat_test;
// 	avg_time_comp3/=repeat_test;
// 	avg_time_post3/=repeat_test;

// 	cout << "[BASELINE]" << endl;
//     cout << "avg_time_parse = " << avg_time_parse1 << endl;
// 	cout << "avg_time_comm = "  << avg_time_comm1 << endl;
// 	cout << "avg_time_comp = "  << avg_time_comp1 << endl;
// 	cout << "avg_time_post = "  << avg_time_post1 << endl;
// 	cout << "total_time = " << avg_time_parse1+avg_time_comm1+avg_time_comp1+avg_time_post1 << endl;

// 	cout << endl;

// 	cout << "[Version 1]" << endl;
// 	cout << "avg_time_parse = " << avg_time_parse2 << endl;
// 	cout << "avg_time_comm = "  << avg_time_comm2 << endl;
// 	cout << "avg_time_comp = "  << avg_time_comp2 << endl;
// 	cout << "avg_time_post = "  << avg_time_post2 << endl;
// 	cout << "total_time = " << avg_time_parse2+avg_time_comm2+avg_time_comp2+avg_time_post2 << endl;

// 	cout << endl;

// 	cout << "[Version2]" << endl;
// 	cout << "avg_time_parse = " << avg_time_parse3 << endl;
// //	cout << "avg_time_comm = "  << avg_time_comm3 << endl;
// 	cout << "avg_time_comp = "  << avg_time_comp3 << endl;
// 	cout << "avg_time_post = "  << avg_time_post3 << endl;
// 	cout << "total_time = " << avg_time_parse3+avg_time_comp3+avg_time_post3 << endl;

}