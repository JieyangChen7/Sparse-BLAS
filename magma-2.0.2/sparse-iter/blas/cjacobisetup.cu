#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date May 2016

       @generated from sparse-iter/blas/zjacobisetup.cu normal z -> c, Mon May  2 23:30:44 2016
       @author Hartwig Anzt

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512


#define PRECISION_c

__global__ void 
cvjacobisetup_gpu(  int num_rows, 
                    int num_vecs,
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *c,
                    magmaFloatComplex *x)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ ){
            c[row+i*num_rows] = b[row+i*num_rows] / d[row];
            x[row+i*num_rows] = c[row+i*num_rows];
        }
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_c_matrix
                RHS b

    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    c           magma_c_matrix*
                c = D^(-1) * b

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_cgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobisetup_vector_gpu(
    magma_int_t num_rows, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix c,
    magma_c_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    int num_vecs = b.num_rows / num_rows;
    magma_int_t threads = BLOCK_SIZE;
    cvjacobisetup_gpu<<< grid, threads, 0, queue->hip_stream()>>>
                ( num_rows, num_vecs, b.dval, d.dval, c.dval, x->val );

    return MAGMA_SUCCESS;
}






__global__ void 
cjacobidiagscal_kernel(  int num_rows,
                         int num_vecs, 
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *c)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
            c[row+i*num_rows] = b[row+i*num_rows] * d[row];
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_c_matrix
                RHS b

    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    c           magma_c_matrix*
                c = D^(-1) * b
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobi_diagscal(
    magma_int_t num_rows, 
    magma_c_matrix d, 
    magma_c_matrix b, 
    magma_c_matrix *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, 512 ));
    int num_vecs = b.num_rows*b.num_cols/num_rows;
    magma_int_t threads = 512;
    cjacobidiagscal_kernel<<< grid, threads, 0, queue->hip_stream()>>>( num_rows, num_vecs, b.dval, d.dval, c->val );

    return MAGMA_SUCCESS;
}













__global__ void 
cjacobiupdate_kernel(  int num_rows,
                       int num_cols, 
                    magmaFloatComplex *t, 
                    magmaFloatComplex *b, 
                    magmaFloatComplex *d, 
                    magmaFloatComplex *x)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < num_rows ){
        for( int i=0; i<num_cols; i++)
            x[row+i*num_rows] += (b[row+i*num_rows]-t[row+i*num_rows]) * d[row];
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-t)
    where d is the diagonal of the system matrix A and t=Ax.

    Arguments
    ---------
                
    @param[in]
    t           magma_c_matrix
                t = A*x
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobiupdate(
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    cjacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );

    return MAGMA_SUCCESS;
}










__global__ void 
cjacobispmvupdate_kernel(  
    int num_rows,
    int num_cols, 
    magmaFloatComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaFloatComplex *t, 
    magmaFloatComplex *b, 
    magmaFloatComplex *d, 
    magmaFloatComplex *x )
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int j;

    if(row<num_rows){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}





/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_c_matrix
                system matrix
                
    @param[in]
    t           magma_c_matrix
                workspace
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobispmvupdate(
    magma_int_t maxiter,
    magma_c_matrix A,
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //magmaFloatComplex c_zero = MAGMA_C_ZERO;
    //magmaFloatComplex c_one = MAGMA_C_ONE;

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_c_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // cjacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        cjacobispmvupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );
    }

    return MAGMA_SUCCESS;
}




__global__ void 
cjacobispmvupdate_bw_kernel(  
    int num_rows,
    int num_cols, 
    magmaFloatComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaFloatComplex *t, 
    magmaFloatComplex *b, 
    magmaFloatComplex *d, 
    magmaFloatComplex *x )
{
    int row_tmp = blockDim.x * blockIdx.x + threadIdx.x;
    int row = num_rows-1 - row_tmp;
    int j;

    if( row>-1 ){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] += (b[row+i*num_rows]-dot) * d[row];
        }
    }
}





/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)
    This kernel processes the thread blocks in reversed order.

    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations   
                
    @param[in]
    A           magma_c_matrix
                system matrix
                
    @param[in]
    t           magma_c_matrix
                workspace
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobispmvupdate_bw(
    magma_int_t maxiter,
    magma_c_matrix A,
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //magmaFloatComplex c_zero = MAGMA_C_ZERO;
    //magmaFloatComplex c_one = MAGMA_C_ONE;

    dim3 grid( magma_ceildiv( t.num_rows, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;

    for( magma_int_t i=0; i<maxiter; i++ ) {
        // distinct routines imply synchronization
        // magma_c_spmv( c_one, A, *x, c_zero, t, queue );                // t =  A * x
        // cjacobiupdate_kernel<<< grid, threads, 0, queue->hip_stream()>>>( t.num_rows, t.num_cols, t.dval, b.dval, d.dval, x->dval );
        // merged in one implies asynchronous update
        cjacobispmvupdate_bw_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval );
    }

    return MAGMA_SUCCESS;
}





__global__ void 
cjacobispmvupdateselect_kernel(  
    int num_rows,
    int num_cols, 
    int num_updates, 
    magma_index_t * indices, 
    magmaFloatComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaFloatComplex *t, 
    magmaFloatComplex *b, 
    magmaFloatComplex *d, 
    magmaFloatComplex *x,
    magmaFloatComplex *y )
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int j;

    if(  idx<num_updates){
        int row = indices[ idx ];
        printf(" ");    
        //if( row < num_rows ){
        magmaFloatComplex dot = MAGMA_C_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( int i=0; i<num_cols; i++){
            for( j=start; j<end; j++){
                dot += dval[ j ] * x[ dcolind[j]+i*num_rows ];
            }
            x[row+i*num_rows] = x[row+i*num_rows] + (b[row+i*num_rows]-dot) * d[row];
            
            //magmaFloatComplex add = (b[row+i*num_rows]-dot) * d[row];
            //#if defined(PRECISION_s) //|| defined(PRECISION_d)
            //    atomicAdd( x + row + i*num_rows, add );  
            //#endif
            // ( unsigned int* address, unsigned int val);
        //}
        }
    }
}


/**
    Purpose
    -------

    Updates the iteration vector x for the Jacobi iteration
    according to
        x=x+d.*(b-Ax)
        
    This kernel allows for overlapping domains: the indices-array contains
    the locations that are updated. Locations may be repeated to simulate
    overlapping domains.


    Arguments
    ---------

    @param[in]
    maxiter     magma_int_t
                number of Jacobi iterations
                
    @param[in]
    num_updates magma_int_t
                number of updates - length of the indices array
                    
    @param[in]
    indices     magma_index_t*
                indices, which entries of x to update
                
    @param[in]
    A           magma_c_matrix
                system matrix
                
    @param[in]
    t           magma_c_matrix
                workspace
                
    @param[in]
    b           magma_c_matrix
                RHS b
                
    @param[in]
    d           magma_c_matrix
                vector with diagonal entries
   
    @param[in]
    tmp         magma_c_matrix
                workspace

    @param[out]
    x           magma_c_matrix*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_c
    ********************************************************************/

extern "C" magma_int_t
magma_cjacobispmvupdateselect(
    magma_int_t maxiter,
    magma_int_t num_updates,
    magma_index_t *indices,
    magma_c_matrix A,
    magma_c_matrix t, 
    magma_c_matrix b, 
    magma_c_matrix d, 
    magma_c_matrix tmp, 
    magma_c_matrix *x,
    magma_queue_t queue )
{
    // local variables
    //magmaFloatComplex c_zero = MAGMA_C_ZERO
    //magmaFloatComplex c_one = MAGMA_C_ONE;
    
    //magma_c_matrix swp;

    dim3 grid( magma_ceildiv( num_updates, BLOCK_SIZE ));
    magma_int_t threads = BLOCK_SIZE;
    printf("num updates:%d %d %d\n", int(num_updates), int(threads), int(grid.x) );

    for( magma_int_t i=0; i<maxiter; i++ ) {
        cjacobispmvupdateselect_kernel<<< grid, threads, 0, queue->hip_stream()>>>
            ( t.num_rows, t.num_cols, num_updates, indices, A.dval, A.drow, A.dcol, t.dval, b.dval, d.dval, x->dval, tmp.dval );
        magma_queue_sync( queue );
        //magma_device_sync();
        //swp.dval = x->dval;
        //x->dval = tmp.dval;
        //tmp.dval = swp.dval;
    }
    
    return MAGMA_SUCCESS;
}
