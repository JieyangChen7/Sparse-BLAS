#include "hip/hip_runtime.h"
/*
   -- MAGMA (version 2.0.2) --
   Univ. of Tennessee, Knoxville
   Univ. of California, Berkeley
   Univ. of Colorado, Denver
   @date May 2016

   @author Azzam Haidar
   @author Tingxing Dong

   @generated from magmablas/zgeqr2_kernels.cu normal z -> c, Mon May  2 23:30:42 2016
*/

#include "magma_internal.h"
#include "batched_kernel_param.h"



__global__ void
cgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magmaFloatComplex **dV_array,    int ldv,
                  magmaFloatComplex **dR_array,    int ldr)
{
    magmaFloatComplex *dV = dV_array[blockIdx.x];
    magmaFloatComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if ( tid < n && column < n) 
    {
        for (int i=column; i < n; i++)
        {
            dR[tid + i * ldr]  =  dV[tid + i * ldv];  
        }
    }
}


/**
    Purpose
    -------
    These are internal routines that might have many assumption.
    They are used in cgeqrf_batched.cpp   

    Copy part of the data in dV to dR
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The order of the matrix .  N >= 0.

    @param[in]
    nb      INTEGER
            Tile size in matrix.  nb <= N.

    @param[in]
    dV_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX array on the GPU, dimension (LDDA,N).

    @param[in]
    lddv    INTEGER
            The leading dimension of each array V.  LDDV >= max(1,N).

    @param[in,out]
    dR_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX array on the GPU, dimension (LDDR,N).

    @param[in]
    lddr    INTEGER
            The leading dimension of each array R.  LDDR >= max(1,N).

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_cgeqrf_aux
    ********************************************************************/

void cgeqrf_copy_upper_batched(                
    magma_int_t n, magma_int_t nb,
    magmaFloatComplex **dV_array, magma_int_t lddv,
    magmaFloatComplex **dR_array, magma_int_t lddr,
    magma_int_t batchCount,
    magma_queue_t queue)
{
    /* 
        copy some data in dV to dR
    */
    if ( nb >= n) return;
    
    cgeqrf_copy_upper_kernel_batched
        <<< batchCount, n, 0, queue->hip_stream() >>>
        ( n, nb, dV_array, lddv, dR_array, lddr );
}
