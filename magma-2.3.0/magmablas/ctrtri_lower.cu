#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.3.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date November 2017

       @generated from magmablas/ztrtri_lower.cu, normal z -> c, Wed Nov 15 00:34:22 2017

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       This file implements lower case, and is called by ctrtri_kernel.cu.
       It's convenient to have separate files for lower & upper, to diff the sources.
*/

#include "magma_internal.h"

#define TRTRI_NONBATCHED
#include "ctrtri.cuh"
#include "ctrtri_lower_device.cuh"


/******************************************************************************/
__global__ void
ctrtri_diag_lower_kernel(
    magma_diag_t diag, int n, const magmaFloatComplex *A, int lda, magmaFloatComplex *d_dinvA)
{
    ctrtri_diag_lower_device(diag, n, A, lda, d_dinvA);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm16_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm16_part2_lower_device( n,  Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm32_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm32_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm64_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part1_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part1_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part2_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part2_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}


/******************************************************************************/
__global__ void
triple_cgemm_above64_part3_lower_kernel(
    int n, const magmaFloatComplex *Ain, int lda, magmaFloatComplex *d_dinvA, int jb, int npages)
{
    triple_cgemm_above64_part3_lower_device( n, Ain, lda, d_dinvA, jb, npages);
}
